#include "hip/hip_runtime.h"
  // Copyright 2011-2012 Justus Sagemüller.

  // This file is part of the Cqtx library.
   //This library is free software: you can redistribute it and/or modify
  // it under the terms of the GNU General Public License as published by
 //  the Free Software Foundation, either version 3 of the License, or
//   (at your option) any later version.
   //This library is distributed in the hope that it will be useful,
  // but WITHOUT ANY WARRANTY; without even the implied warranty of
 //  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.
  // You should have received a copy of the GNU General Public License
 //  along with this library.  If not, see <http://www.gnu.org/licenses/>.


#include "squdistaccel-fns.hcu"
#include "hipblas.h"
#include <math.h>
#include <assert.h>
#include <stdio.h>


__global__ void ca_d_gaussian_var_x0_invtwosigmasq_A
                ( const double* x
                , double x0
                , double inv_twosigmasq
                , double A
                , const double* rcmp
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    double deltax = x[i] - x0;
    resc[i] = rcmp[i] - A*exp(-deltax*deltax*inv_twosigmasq);
  }
}


double cudaaccelsqd_gaussian_VAR_x0_sigma_A
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  static int block_size = 256;
  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_gaussian_var_x0_invtwosigmasq_A<<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fitparams[0]
                         , 1./(2.*fitparams[1]*fitparams[1])
                         , fitparams[2]
                         , fixed->measurevalseqs[0]
                         , fixed->fnresbuffer
                         , fixed->n_measures               );
  double result;

  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );
  return result*result;  //nrm2 returns √(∑ x²).
}


__global__ void ca_d_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_RET
                ( const double* x
                , double x0
                , double inv_twosigmasq
                , double A
                , const double* rcmp
                , const double* rcmpuncrt
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    double deltax = x[i] - x0;
    resc[i] = (rcmp[i] - A*exp(-deltax*deltax*inv_twosigmasq))
                                   / rcmpuncrt[i];
  }
}

double cudaaccelsqd_gaussian_VAR_x0_sigma_A_UNCRT_RET
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  static int block_size = 256;
  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_RET<<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fitparams[0]
                         , 1./(2.*fitparams[1]*fitparams[1])
                         , fitparams[2]
                         , fixed->measurevalseqs[0]
                         , fixed->measurevalseqs[2]
                         , fixed->fnresbuffer
                         , fixed->n_measures               );
  double result;

  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );
  return result*result;  //nrm2 returns √(∑ x²).
}


__global__ void ca_d_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_x
                ( const double* x
                , const double* xuncrt
                , double x0
                , double inv_twosigmasq
                , double A
                , const double* rcmp
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    double deltax = x[i] - x0
         , exp_argcoeff = -deltax*inv_twosigmasq
         , expf_res = A*exp(deltax*exp_argcoeff)
         , expf_uncrt = xuncrt[i] * expf_res * 2 * exp_argcoeff;
    resc[i] = (rcmp[i] - expf_res) / expf_uncrt;
  }
}

double cudaaccelsqd_gaussian_VAR_x0_sigma_A_UNCRT_x
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  static int block_size = 256;
  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_x<<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fixed->measurevalseqs[2]
                         , fitparams[0]
                         , 1./(2.*fitparams[1]*fitparams[1])
                         , fitparams[2]
                         , fixed->measurevalseqs[0]
                         , fixed->fnresbuffer
                         , fixed->n_measures               );
  double result;

  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );
  return result*result;  //nrm2 returns √(∑ x²).
}


__global__ void ca_sqd_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_RET_x
                ( const double* x
                , const double* xuncrt
                , double x0
                , double inv_twosigmasq
                , double A
                , const double* rcmp
                , const double* rcmpuncrt
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    double deltax = x[i] - x0
         , exp_argcoeff = -deltax*inv_twosigmasq
         , expf_res = A*exp(deltax*exp_argcoeff)
         , expf_uncrt = xuncrt[i] * expf_res * 2 * exp_argcoeff
         , deltaret = rcmp[i] - expf_res;
    resc[i] = deltaret*deltaret
                / (expf_uncrt*expf_uncrt + rcmpuncrt[i]*rcmpuncrt[i]);
  }
}

double cudaaccelsqd_gaussian_VAR_x0_sigma_A_UNCRT_RET_x
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  static int block_size = 256;
  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_sqd_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_RET_x<<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fixed->measurevalseqs[3]
                         , fitparams[0]
                         , 1./(2.*fitparams[1]*fitparams[1])
                         , fitparams[2]
                         , fixed->measurevalseqs[0]
                         , fixed->measurevalseqs[2]
                         , fixed->fnresbuffer
                         , fixed->n_measures                 );
  double result;

  fixed->cublasstat
     = hipblasDasum( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );
  return result;
}




template<unsigned NPeaks>
struct multigaussian_VAR_x0_invtwosigmasq_A_PARAMS {
  double x0[NPeaks], inv_twosigmasq[NPeaks], A[NPeaks];
};

template<>
struct multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<0> {
  union { double* x0; double* inv_twosigmasq; double* A; };
};

template<unsigned NPeaks>
__global__ void ca_d_multigaussian_VARS_x0_invtwosigmasq_A
                ( const double* x
                , multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> ps
                , const double* rcmp
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  //static_assert(NPeaks>0);
  if(i<N) {
    double fnres = 0;
    for(unsigned j=0; j<NPeaks; ++j) {
      double deltax = x[i] - ps.x0[j];
      fnres += ps.A[j] * exp(-deltax*deltax * ps.inv_twosigmasq[j]);
    }
    resc[i] = rcmp[i] - fnres;
  }
}


template<unsigned NPeaks>
double cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
/*  double* peakprev = (double*) malloc(fixed->n_measures * sizeof(double));
  double* peakprev1 = (double*) malloc(fixed->n_measures * sizeof(double));

  printf("Create arg tgt preview, %d values...\n", fixed->n_measures);
  hipblasGetVector(fixed->n_measures, sizeof(double), fixed->measurevalseqs[0], 1, peakprev, 1);
  hipblasGetVector(fixed->n_measures, sizeof(double), fixed->measurevalseqs[1], 1, peakprev1, 1);
  for(int k=0; k<fixed->n_measures; ++k) printf("%f\t @ %f\n", peakprev[k], peakprev1[k]);
*/
  static int block_size = 256;

  multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> peaksargs;

  for(unsigned j=0; j<NPeaks; ++j) {
    peaksargs.x0[j] = fitparams[0 + 3*j];
    peaksargs.inv_twosigmasq[j] = 1./(2.*fitparams[1 + 3*j]*fitparams[1 + 3*j]);
    peaksargs.A[j] = fitparams[2 + 3*j];
  }

  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_multigaussian_VARS_x0_invtwosigmasq_A<NPeaks>
            <<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , peaksargs
                         , fixed->measurevalseqs[0]
                         , fixed->fnresbuffer
                         , fixed->n_measures        );
  double result;
/*
  printf("Create fn result preview, %d values...\n", fixed->n_measures);
  hipblasGetVector(fixed->n_measures, sizeof(double), fixed->fnresbuffer, 1, peakprev, 1);
  for(int k=0; k<fixed->n_measures; ++k) printf("%f\n", peakprev[k]);
*/
  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );

//  free(peakprev);  free(peakprev1);

  return result*result;  //nrm2 returns √(∑ x²).
}

template<>
double cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<0>
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  double result;
  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->measurevalseqs[0]
                  , 1
                  , &result              );
  return result*result;  //nrm2 returns √(∑ x²).
}



#define MANY_SQDF_TEMPLATE_INSTANTIATIONS(tf)                                                                                                  \
const cudaNonlinSqdistEvalFunction tf##_vtable[]                                                                                               \
        = { tf<0>, tf<1>, tf<2>, tf<3>, tf<4>, tf<5>, tf<6>, tf<7>, tf<8>, tf<9>, tf<10>, tf<11>, tf<12>, tf<13>, tf<14>, tf<15>               \
          , tf<16>, tf<17>, tf<18>, tf<19>, tf<20>, tf<21>, tf<22>, tf<23>, tf<24>, tf<25>, tf<26>, tf<27>, tf<28>, tf<29>, tf<30>, tf<31> };  \
const unsigned n_instantiated_##tf = 32;


MANY_SQDF_TEMPLATE_INSTANTIATIONS(cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A)

cudaNonlinSqdistEvalFunction cudaaccelsqd_multigaussian_VARS_x0_sigma_A
              ( unsigned npeaks ) {
  if(npeaks < n_instantiated_cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A)
    return cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_vtable[npeaks];
   else
    return NULL;
}




template<unsigned NPeaks>
__global__ void ca_d_multigaussian_VARS_x0_invtwosigmasq_A_UNCRT_RET
                ( const double* x
                , multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> ps
                , const double* rcmp
                , const double* rcmpuncrt
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  //static_assert(NPeaks>0);
  if(i<N) {
    double fnres = 0, xi=x[i];
    for(unsigned j=0; j<NPeaks; ++j) {
      double deltax = xi - ps.x0[j];
      fnres += ps.A[j] * exp(-deltax*deltax * ps.inv_twosigmasq[j]);
    }
    resc[i] = (rcmp[i] - fnres)/rcmpuncrt[i];
  }
}


template<unsigned NPeaks>
double cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {

  static int block_size = 256;

  multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> peaksargs;

  for(unsigned j=0; j<NPeaks; ++j) {
    peaksargs.x0[j] = fitparams[0 + 3*j];
    peaksargs.inv_twosigmasq[j] = 1./(2.*fitparams[1 + 3*j]*fitparams[1 + 3*j]);
    peaksargs.A[j] = fitparams[2 + 3*j];
  }

  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_multigaussian_VARS_x0_invtwosigmasq_A_UNCRT_RET<NPeaks>
            <<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , peaksargs
                         , fixed->measurevalseqs[0]
                         , fixed->measurevalseqs[2]
                         , fixed->fnresbuffer
                         , fixed->n_measures        );
  double result;

  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );

  return result*result;  //nrm2 returns √(∑ x²).
}


MANY_SQDF_TEMPLATE_INSTANTIATIONS(cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET)


cudaNonlinSqdistEvalFunction cudaaccelsqd_multigaussian_VARS_x0_sigma_A_UNCRT_RET
              ( unsigned npeaks ) {
  if(npeaks < n_instantiated_cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET)
    return cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET_vtable[npeaks];
   else
    return NULL;
}




template<unsigned NPeaks>
__global__ void ca_sqd_multigaussian_VARS_x0_invtwosigmasq_A_UNCRT_x
                ( const double* x
                , const double* xuncrt
                , multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> ps
                , const double* rcmp
                , double* resc
                , int N                                                  ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  //static_assert(NPeaks>0);
  if(i<N) {
    double fnres = 0, fnderivative = 0, xi=x[i];

    for(unsigned j=0; j<NPeaks; ++j) {
      double deltax = xi - ps.x0[j]
           , exp_argcoeff = -deltax*ps.inv_twosigmasq[j]
           , expf_res = ps.A[j]*exp(deltax*exp_argcoeff);

      fnres += expf_res;
      fnderivative += expf_res * 2 * exp_argcoeff;
    }
    double deltaret = rcmp[i] - fnres
         , resuncrt = xuncrt[i] * fnderivative;
    resc[i] = deltaret * deltaret
                 / ( resuncrt*resuncrt );
  }
}


template<unsigned NPeaks>
double cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_x
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  static int block_size = 256;

  multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> peaksargs;

  for(unsigned j=0; j<NPeaks; ++j) {
    peaksargs.x0[j] = fitparams[0 + 3*j];
    peaksargs.inv_twosigmasq[j] = 1./(2.*fitparams[1 + 3*j]*fitparams[1 + 3*j]);
    peaksargs.A[j] = fitparams[2 + 3*j];
  }

  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_sqd_multigaussian_VARS_x0_invtwosigmasq_A_UNCRT_x<NPeaks>
            <<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fixed->measurevalseqs[2]
                         , peaksargs
                         , fixed->measurevalseqs[0]
                         , fixed->fnresbuffer
                         , fixed->n_measures        );
  double result;

  fixed->cublasstat
     = hipblasDasum( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );

  return result;
}


MANY_SQDF_TEMPLATE_INSTANTIATIONS(cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_x)


cudaNonlinSqdistEvalFunction cudaaccelsqd_multigaussian_VARS_x0_sigma_A_UNCRT_x
              ( unsigned npeaks ) {
  if(npeaks < n_instantiated_cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_x)
    return cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_x_vtable[npeaks];
   else
    return NULL;
}



template<unsigned NPeaks>
__global__ void ca_sqd_multigaussian_VARS_x0_invtwosigmasq_A_UNCRT_RET_x
                ( const double* x
                , const double* xuncrt
                , multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> ps
                , const double* rcmp
                , const double* rcmpuncrt
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  //static_assert(NPeaks>0);
  if(i<N) {
    double fnres = 0, fnderivative = 0, xi=x[i];

    for(unsigned j=0; j<NPeaks; ++j) {
      double deltax = xi - ps.x0[j]
           , exp_argcoeff = -deltax*ps.inv_twosigmasq[j]
           , expf_res = ps.A[j]*exp(deltax*exp_argcoeff);

      fnres += expf_res;
      fnderivative += expf_res * 2 * exp_argcoeff;
         // ∂ₓ A⋅exp(-x²/(2σ²)) = Aexp(-x²/(2σ²))⋅2⋅(-x/(2σ²))
    }

    double deltaret = rcmp[i] - fnres
         , resuncrt = xuncrt[i] * fnderivative;
    resc[i] = deltaret * deltaret
                 / ( resuncrt*resuncrt + rcmpuncrt[i]*rcmpuncrt[i] );
  }
}


template<unsigned NPeaks>
double cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET_x
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {

  static int block_size = 256;

  multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> peaksargs;

  for(unsigned j=0; j<NPeaks; ++j) {
    peaksargs.x0[j] = fitparams[0 + 3*j];
    peaksargs.inv_twosigmasq[j] = 1./(2.*fitparams[1 + 3*j]*fitparams[1 + 3*j]);
    peaksargs.A[j] = fitparams[2 + 3*j];
  }

  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_sqd_multigaussian_VARS_x0_invtwosigmasq_A_UNCRT_RET_x<NPeaks>
            <<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fixed->measurevalseqs[3]
                         , peaksargs
                         , fixed->measurevalseqs[0]
                         , fixed->measurevalseqs[2]
                         , fixed->fnresbuffer
                         , fixed->n_measures        );
  double result;

  fixed->cublasstat
     = hipblasDasum( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );

  return result;
}

MANY_SQDF_TEMPLATE_INSTANTIATIONS(cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET_x)


cudaNonlinSqdistEvalFunction cudaaccelsqd_multigaussian_VARS_x0_sigma_A_UNCRT_RET_x
              ( unsigned npeaks ) {
  if(npeaks < n_instantiated_cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET_x)
    return cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_UNCRT_RET_x_vtable[npeaks];
   else
    return NULL;
}
