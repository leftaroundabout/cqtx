#include "hip/hip_runtime.h"
  // Copyright 2011-2012 Justus Sagemüller.

  // This file is part of the Cqtx library.
   //This library is free software: you can redistribute it and/or modify
  // it under the terms of the GNU General Public License as published by
 //  the Free Software Foundation, either version 3 of the License, or
//   (at your option) any later version.
   //This library is distributed in the hope that it will be useful,
  // but WITHOUT ANY WARRANTY; without even the implied warranty of
 //  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
//   GNU General Public License for more details.
  // You should have received a copy of the GNU General Public License
 //  along with this library.  If not, see <http://www.gnu.org/licenses/>.


#include "squdistaccel-fns.hcu"
#include "hipblas.h"
#include <math.h>
#include <assert.h>
#include <stdio.h>


__global__ void ca_d_gaussian_var_x0_invtwosigmasq_A
                ( const double* x
                , double x0
                , double inv_twosigmasq
                , double A
                , const double* rcmp
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    double deltax = x[i] - x0;
    resc[i] = rcmp[i] - A*exp(-deltax*deltax*inv_twosigmasq);
  }
}


double cudaaccel_sqdf_gaussian_var_x0_sigma_A
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  static int block_size = 256;
  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_gaussian_var_x0_invtwosigmasq_A<<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fitparams[0]
                         , 1./(2.*fitparams[1]*fitparams[1])
                         , fitparams[2]
                         , fixed->measurevalseqs[0]
                         , fixed->fnresbuffer
                         , fixed->n_measures               );
  double result;

  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );
  return result*result;  //nrm2 returns √(∑ x²).
}



template<unsigned NPeaks>
struct multigaussian_VAR_x0_invtwosigmasq_A_PARAMS {
  double x0[NPeaks], inv_twosigmasq[NPeaks], A[NPeaks];
};

template<unsigned NPeaks>
__global__ void ca_d_multigaussian_VARS_x0_invtwosigmasq_A
                ( const double* x
                , multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> ps
                , const double* rcmp
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  //static_assert(NPeaks>0);
  if(i<N) {
    double fnres = 0;
    for(unsigned j=0; j<NPeaks; ++j) {
      double deltax = x[i] - ps.x0[j];
      fnres += ps.A[j] * exp(-deltax*deltax * ps.inv_twosigmasq[j]);
    }
    resc[i] = rcmp[i] - fnres;
  }
}


template<unsigned NPeaks>
double cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
/*  double* peakprev = (double*) malloc(fixed->n_measures * sizeof(double));
  double* peakprev1 = (double*) malloc(fixed->n_measures * sizeof(double));

  printf("Create arg tgt preview, %d values...\n", fixed->n_measures);
  hipblasGetVector(fixed->n_measures, sizeof(double), fixed->measurevalseqs[0], 1, peakprev, 1);
  hipblasGetVector(fixed->n_measures, sizeof(double), fixed->measurevalseqs[1], 1, peakprev1, 1);
  for(int k=0; k<fixed->n_measures; ++k) printf("%f\t @ %f\n", peakprev[k], peakprev1[k]);
*/
  static int block_size = 256;

  multigaussian_VAR_x0_invtwosigmasq_A_PARAMS<NPeaks> peaksargs;

  for(unsigned j=0; j<NPeaks; ++j) {
    peaksargs.x0[j] = fitparams[0 + 3*j];
    peaksargs.inv_twosigmasq[j] = 1./(2.*fitparams[1 + 3*j]*fitparams[1 + 3*j]);
    peaksargs.A[j] = fitparams[2 + 3*j];
  }

  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_multigaussian_VARS_x0_invtwosigmasq_A<NPeaks>
            <<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , peaksargs
                         , fixed->measurevalseqs[0]
                         , fixed->fnresbuffer
                         , fixed->n_measures        );
  double result;
/*
  printf("Create fn result preview, %d values...\n", fixed->n_measures);
  hipblasGetVector(fixed->n_measures, sizeof(double), fixed->fnresbuffer, 1, peakprev, 1);
  for(int k=0; k<fixed->n_measures; ++k) printf("%f\n", peakprev[k]);
*/
  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );

//  free(peakprev);  free(peakprev1);

  return result*result;  //nrm2 returns √(∑ x²).
}

template<>
double cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<0>
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  double result;
  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->measurevalseqs[0]
                  , 1
                  , &result              );
  return result*result;  //nrm2 returns √(∑ x²).
}


const cudaNonlinSqdistEvalFunction
   cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_vtable[]
        = { cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<0>
//          , cudaaccel_sqdf_gaussian_VAR_x0_sigma_A
          , cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<1>
          , cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<2>
          , cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<3>  /*
          , ...                                        ... */ , cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<4>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<5>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<6>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<7>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<8>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<9>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<10>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<11>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<12>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<13>, cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<14>
          , cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A<15>  };
const unsigned n_instantiated_cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A
                                                       = 16;

cudaNonlinSqdistEvalFunction cudaaccel_multigaussian_VARS_x0_sigma_A
              ( unsigned npeaks ) {
  if(npeaks < n_instantiated_cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A)
    return cudaaccel_sqdf_multigaussian_VARS_x0_sigma_A_vtable[npeaks];
   else
    return NULL;
}






__global__ void ca_d_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_RET
                ( const double* x
                , double x0
                , double inv_twosigmasq
                , double A
                , const double* rcmp
                , const double* rcmpuncrt
                , double* resc
                , int N                    ) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i<N) {
    double deltax = x[i] - x0;
    resc[i] = (rcmp[i] - A*exp(-deltax*deltax*inv_twosigmasq))
                                   / rcmpuncrt[i];
  }
}

double cudaaccel_sqdf_gaussian_var_x0_sigma_A_UNCRT_RET
                ( cudastoredMeasureseqRepHandle* fixed
                , const double* fitparams              ) {
  static int block_size = 256;
  int n_blocks = fixed->n_measures/block_size
              + (fixed->n_measures%block_size==0? 0 : 1);
  ca_d_gaussian_VAR_x0_invtwosigmasq_A_UNCRT_RET<<<n_blocks, block_size>>>
                         ( fixed->measurevalseqs[1]
                         , fitparams[0]
                         , 1./(2.*fitparams[1]*fitparams[1])
                         , fitparams[2]
                         , fixed->measurevalseqs[0]
                         , fixed->measurevalseqs[2]
                         , fixed->fnresbuffer
                         , fixed->n_measures               );
  double result;

  fixed->cublasstat
     = hipblasDnrm2( *fixed->cublashandle
                  , fixed->n_measures
                  , fixed->fnresbuffer
                  , 1
                  , &result              );
  return result*result;  //nrm2 returns √(∑ x²).
}

